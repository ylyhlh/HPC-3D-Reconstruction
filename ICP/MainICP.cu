
#include <hip/hip_runtime.h>
#include <stdio.h>

#define blocksizex 16 
#define blocksizey 16 
#define N 40000
#define DOT(a,b) a[0]*b[0]+a[1]*b[1]+a[2]*b[2]
#define DOTVA(a,b) a.x*b[0]+a.y*b[1]+a.z*b[2]
 typedef struct 
 {
   float x;
   float y;
   float z;
 } vertex;
  typedef struct 
 {
   vertex a;
   vertex n;
 } mata;
 
__global__ void RunICP(
int   validPoints[N],
vertex verProjD[N],
vertex verNormD[N],
float matRgPre[9],
float matTransltgPre[3],
const int focus,
vertex verProS[N],
vertex verNormS[N],
const float epsD,
const float epsN,
const float epsDs,
float Energy[N],
mata Ak[N],
float Bk[N],
int flag[N]
)
{
	int gidx=blockIdx.x*blockDim.x+threadIdx.x;
        int gidy=blockIdx.y*blockDim.y+threadIdx.y;
        int i=gidx+(gidy-1)*gridDim.y;//num of order
        float vertexProj[3];
        float normalProj[3];
        float disVdiff[3];
        float disNdiff[3];
        float error;

		if (validPoints[i] == true && i<N){
			//transformV  and trandformR
            
			vertexProj[0] = verProjD[i].x*matRgPre[0]+verProjD[i].y*matRgPre[1]+verProjD[i].z*matRgPre[2]+matTransltgPre[0];
			vertexProj[1] = verProjD[i].x*matRgPre[3]+verProjD[i].y*matRgPre[4]+verProjD[i].z*matRgPre[5]+matTransltgPre[1];
			vertexProj[2] = verProjD[i].x*matRgPre[6]+verProjD[i].y*matRgPre[7]+verProjD[i].z*matRgPre[8]+matTransltgPre[2];
			
			normalProj[0] = verNormD[i].x*matRgPre[0]+verNormD[i].y*matRgPre[1]+verNormD[i].z*matRgPre[2];
			normalProj[1] = verProjD[i].x*matRgPre[3]+verProjD[i].y*matRgPre[4]+verProjD[i].z*matRgPre[5];
			normalProj[2] = verProjD[i].x*matRgPre[6]+verProjD[i].y*matRgPre[7]+verProjD[i].z*matRgPre[8];
            

			//get index
			int x = (vertexProj[0] / vertexProj[2] * focus) + 320;

	        int y = (vertexProj[1] / vertexProj[2] * focus) + 240;

	        int index=y*640 + x;

			if(index < 0 || index >= N){

                                Energy[i]=0;
				flag[i]=0;
				return;

			}
			
			disVdiff[0]  = verProS[index].x-vertexProj[0];
			disNdiff[0]  = verNormS[index].x-normalProj[0];
		    disVdiff[1]  = verProS[index].y-vertexProj[1];
			disNdiff[1]  = verNormS[index].y-normalProj[1];
			disVdiff[2]  = verProS[index].y-vertexProj[2];
			disNdiff[2]  = verNormS[index].y-normalProj[2];


			float disV  = DOT(disVdiff, disVdiff);
			float disN  = DOT(disNdiff, disNdiff);

			if( disV < epsD && disN >= epsN && disV > epsDs){
				
				error=DOTVA(verNormS[index],disVdiff);

				Energy[i]= error*error/10;//remember to reduce sumEngry

  				Ak[i].a.x=vertexProj[1]*verNormS[index].z-vertexProj[2]*verNormS[index].y;
                Ak[i].a.y=vertexProj[2]*verNormS[index].x-vertexProj[0]*verNormS[index].z;
                Ak[i].a.z=vertexProj[0]*verNormS[index].y-vertexProj[1]*verNormS[index].x;         

				Ak[i].n.x=verNormS[index].x;
				Ak[i].n.x=verNormS[index].y;
				Ak[i].n.x=verNormS[index].z;

				Bk[i]=DOTVA(verNormS[index],disVdiff);
				flag[i]=1;
			}
           else
                            flag[i]=0;

	     }
             else
             {flag[i]=0;}
}
int main()
{
   //kernel invocation with N threads
   
    /*
   	int   validPoints[N];
   	float verProjD[N][3];
	float verNormD[N][3];
	float matRgPre[3];
	float matTransltgPre[3];

	float verProS[N][3];
	float verNormS[N][3];

	float Energy[N];
	float Ak[N][6];
	float Bk[N];
	int flag[N];
	*/
	const int focus=550;
	const float epsD=40000.0;
	const float epsN=0.6;
	const float epsDs=100.0;
   	//----------------------------------
	//allocation for host
        //------------------------------------
    size_t flsizeN =N*sizeof(float);
 	size_t flsize3N =3*N*sizeof(float);
	size_t flsize3 =3*sizeof(float);
        size_t flsize6N=6*N*sizeof(float);

	int* validPoints=(int*)malloc(flsizeN);
    vertex* verProjD=(vertex*)malloc(flsize3N);
	vertex* verNormD=(vertex*)malloc(flsize3N);
	float* matRgPre=(float*)malloc(9*sizeof(float));
	float* matTransltgPre=(float*)malloc(flsize3);

	vertex* verProS=(vertex*)malloc(flsize3N);
	vertex* verNormS=(vertex*)malloc(flsize3N);

	float* Energy=(float*)malloc(flsizeN);
	mata* Ak=(mata*)malloc(flsize6N);
	float* Bk=(float*)malloc(flsizeN);
	int* flag=(int*)malloc(flsizeN);

	if (validPoints == NULL || verProjD == NULL || verNormD == NULL || 
        matRgPre == NULL||matTransltgPre==NULL||verProS==NULL||verNormS==NULL
       ||Energy==NULL||Ak==NULL||Bk==NULL||flag==NULL)
      {
       fprintf(stderr, "Failed to allocate host vectors!\n");
       exit(EXIT_FAILURE);
      }
      
      //-----------------------------------
      //allocation for device
      //-----------------------------------
        hipError_t err=hipSuccess;
         
        int* d_validPoints=NULL;
        err=hipMalloc((void **)&d_validPoints,flsizeN);
        
        if (err != hipSuccess)
       {
        fprintf(stderr, "Failed to allocate device vector d_validPoints (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
        }


	vertex* d_verProjD=NULL;
        err=hipMalloc((void **)&d_verProjD,flsize3N);
        
        if (err != hipSuccess)
       {
        fprintf(stderr, "Failed to allocate device d_verProjD (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
        }


	vertex* d_verNormD=NULL;
        err=hipMalloc((void **)&d_verNormD,flsize3N);
        
        if (err != hipSuccess)
       {
        fprintf(stderr, "Failed to allocate device d_verNormD (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
        }

	float* d_matRgPre=NULL;
        err=hipMalloc((void **)&d_matRgPre,9*sizeof(float));
        
        if (err != hipSuccess)
       {
        fprintf(stderr, "Failed to allocate device d_matRgPre (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
        }


        float* d_matTransltgPre=NULL;
        err=hipMalloc((void **)&d_matTransltgPre,flsize3);
        
        if (err != hipSuccess)
       {
        fprintf(stderr, "Failed to allocate device d_matTransltgPre (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
        }

	
	vertex* d_verProS=NULL;
        err=hipMalloc((void **)&d_verProS,flsize3N);
        
        if (err != hipSuccess)
       {
        fprintf(stderr, "Failed to allocate device d_verProS (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
        }

   	vertex* d_verNormS=NULL;
        err=hipMalloc((void **)&d_verNormS,flsize3N);
        
        if (err != hipSuccess)
       {
        fprintf(stderr, "Failed to allocate device d_verNormS (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
        }

	
	float* d_Energy=NULL;
        err=hipMalloc((void **)&d_Energy,flsizeN);
        
        if (err != hipSuccess)
       {
        fprintf(stderr, "Failed to allocate device d_Energy (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
        }


	mata* d_Ak=NULL;
        err=hipMalloc((void **)&d_Ak,flsize6N);
        
        if (err != hipSuccess)
       {
        fprintf(stderr, "Failed to allocate device d_Ak (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
        }

	
	float* d_Bk=NULL;
        err=hipMalloc((void **)&d_Bk,flsizeN);
        
        if (err != hipSuccess)
       {
        fprintf(stderr, "Failed to allocate device d_Bk (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
        }

	int* d_flag=NULL;
        err=hipMalloc((void **)&d_flag,flsizeN);
        
        if (err != hipSuccess)
       {
        fprintf(stderr, "Failed to allocate device d_flag (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
        }

     //--------------------------------------------------------------------------
    // Copy the host input variables in host memory 
    //to the device input  variables in device memory
    //--------------------------------------------------------------------------
    err = hipMemcpy(d_validPoints, validPoints, flsizeN,hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy variables from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_verProjD, verProjD, flsize3N,hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy variables from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_verNormD, verNormD, flsize3N,hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy variables from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_matRgPre, matRgPre, 9*sizeof(float),hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy variables from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_matTransltgPre, matTransltgPre, flsize3,hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy variables from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_verProS, verProS, flsize3N,hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy variables from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_verNormS, verNormS, flsize3N,hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy variables from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    //output ----------------------


    

   dim3 threadsPerBlock(blocksizex,blocksizey);
   dim3 numBlocks(N/threadsPerBlock.x+1,N/threadsPerBlock.y+1);

   RunICP<<<numBlocks,threadsPerBlock>>>(
   d_validPoints,
   d_verProjD,
   d_verNormD,
   d_matRgPre,
   d_matTransltgPre,
   focus,
   d_verProS,
   d_verNormS,
   epsD,
   epsN,
   epsDs,
   d_Energy,
   d_Ak,
   d_Bk,
   d_flag
   );

   //--------------------------------------------------------------------------
    // Copy the device result variables in device memory to the host result variables
    // in host memory.
    //--------------------------------------------------------------------------
    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(Energy, d_Energy, flsizeN, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy variable cube from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
	
    err = hipMemcpy(Ak, d_Ak, flsize6N, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy variable cube from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(Bk, d_Bk, flsizeN, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy variable cube from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(flag, d_flag, flsizeN, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy variable cube from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }



    int sumEnergy=0;
     for(int i=0;i<N;i++)
     {
      if(flag[i]==1)
      {
      
       sumEnergy+=Energy[i];
      //Ak.push_back(d_Ak[i]);

      //Bk.push_back(d_Bk[i]);

      }

     }
}
