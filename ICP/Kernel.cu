#include "hip/hip_runtime.h"
#define dotPro(a,b) a(0)*b(0)+a(1)*b(1)+a(2)*b(2)

__global__ void RunICP(
int   validPoints[N],
float verProjD[N][3],
float verNormD[N][3],
float matRgPre[3],
float matTransltgPre[3],
const int focus,
float verProS[N][3],
float verNormS[N][3],
const float epsD,
const float epsN,
const float epsDs,
float Energy[N],
float Ak[N][6],
float Bk[N],
int flag[N]
)
{
	int gidx=blockIdx.x*blockDim.x+threadIdx.x;
        int gidy=blockIdx.y*blockDim.y+threadIdx.y;
        int i=gidx+(gidy-1)*gridDim.y;//num of order
        float vertexProj[3];
        float normalProj[3];
        float disVdiff[3];
        float disNdiff[3];
        float error;

		if (validPoints[i] == true && i<N){
			//transformV  and trandformR
                       for(int k=1;k<3;k++)
			{
			vertexProj[k] = dotPro(verProjD[i], matRgPre)+matTransltgPre[k];
			
                        normalProj[k] = dotPro(verNormD[i], matRgPre);
			}

			//get index
			int x = (vertexProj[0] / vertexProj[2] * focus) + 320;

	                int y = (vertexProj[1] / vertexProj[2] * focus) + 240;

	                index=y*640 + x;

			if(index < 0 || index >= npts){

                                Energy[i]=0;
				flag[i]=0;
				return;

			}
			
                        for(k=0;k<3;k++){

			disVdiff[k]  = verProjS[index][k]-vertexProj[k];
			disNdiff[k]  = verNormS[index][k]-normalProj[k];

                        }

			disV  = dotPro(disVdiff, disVdiff);
			disN  = dotPro(disNdiff, disNdiff);

			if( disV < epsD && disN >= epsN && disV > epsDs){
				
				error=dotPro(disVdiff,verNormS[index]);

				Energy= error*error/10;//remember to reduce sumEngry

  				Ak[i][0]=vertexProj[1]*verNormS[index][2]-vertexProj[2]*verNormS[index][1];
                                Ak[i][1]=vertexProj[2]*verNormS[index][0]-vertexProj[0]*verNormS[index][2];
                                Ak[i][2]=vertexProj[0]*verNormS[index][1]-vertexProj[1]*verNormS[index][0];
                                
 				for(int k=0;k<3;k++)
                               {
                                
				Ak[i][k+3]=verNormS[index][k];

				Bk[i][k]=dotPro(verNormS[index],disVdiff);
				}
				flag[i]=1;
			}
                        else
                            flag[i]=0;

	     }
             else
             {flag[i]=0}
}
