#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>


//#define POS3(p) p[1]+size_x*p[2]+size_x*size_y*p[3]
#define POS3(i,j,k) k+size_z*j+size_z*size_y*i
#define POS(n,x,y) x*n+y
//location on Rk 
#define DMAP(a) (a[0]+map_center_x)*map_size_y+(a[1]+map_center_y)
// For the CUDA runtime routines (prefixed with "cuda_")

#define DOT(a,b) a[0]*b[0]+a[1]*b[1]+a[2]*b[2]
#define DOTV(a,b) a.x*b.x+a.y*b.y+a.z*b.z

//---------------------------------------------------------------------------
// Some paramenters------------->Define them
//---------------------------------------------------------------------------
//----- paramenters
   #define mu 1.0f//trunction length
   #define weightOweight 1.0f // W_Rk =weightOweight * cos(theta)/R_k
   #define W_eta 100.0f //trunction weight
// Constant
//----- in cube
   //unit length
   #define unit 0.1f
   // Size of cube
   #define size_x 10
   #define size_y 10
   #define size_z 10
   // The index of original point
   #define center_x 5
   #define center_y 5
   #define center_z 5
//----- in depth map
   // Size of depth map-------Define it
   #define map_size_x 9
   #define map_size_y 9
   // The index of original point-------Define it
   #define map_center_x 4
   #define map_center_y 4


//debug
#define numElements size_x*size_y*size_z


//---------------------------vertex
 typedef struct 
 {
   float x;
   float y;
   float z;
 } vertex;

/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
__global__ void
TSDF(float *cube, float *cube_w, float *cube_k, float *cube_wk, float *Rgk, float *tgk, float *Rk, vertex *NRk)
{

    int i = blockDim.x * blockIdx.x + threadIdx.x;
    float a[3]={1,2,3};
    vertex b={.x=1,.y=2,.z=3};
    if (i < numElements )
    {
        cube[i] = cube[i] + cube[i];
        cube[i] = DOTV(b,b);
        
    }/*
    
            int i = 0;
            int j = 0;
            int k = 0;
            
            //p---?? unit
            float p[3], pt[3];
            int x[2];
            p[0] = (i - center_x) *unit;
            p[1] = (j - center_y) *unit;
            p[2] = (k - center_z) *unit;
            //printf("p[2]=%f\n",p[2]);
            
            //T_gk^-1 * p
            pt[0]=p[0]-tgk[0];
            pt[1]=p[1]-tgk[1];
            pt[2]=p[2]-tgk[2];
            
            for ( int s=0 ; s<3 ; s++  )
               for ( int t=0 ; t<3 ; t++  )
               {
                  p[s] += Rgk[ POS(3,t,s) ] * pt[t];
               }
            
            //first part of eta
            float eta=0.0f;
            eta=sqrt(pt[0]*pt[0]+pt[1]*pt[1]+pt[2]*pt[2]);
            
            //pi( K * (T_gk^-1 * p) )
            //round to nearest pixel
            for ( int s=0 ; s<2 ; s++  )
              x[s]=int( p[s] + p[2] + 0.5f );
              
            if(-1 < x[0] < map_size_x && -1< x[1] < map_size_y)
            {
               //lambd=norm(K^-1*x)
               float lambd = 0.0f;
               for ( int s=0 ; s<2 ; s++  )
                  lambd += x[s] * x[s];

               lambd=sqrt( lambd + 1.0f );
               

               //second part of eta
               eta=eta / lambd + Rk[DMAP(x)];
               
               
               //F_Rk(p)
               if (eta>=-mu)
                  cube_k[ POS3(i,j,k) ] = min(1.0f,eta/mu)*(-1);
               //W_Rk(p)
               cube_wk[ POS3(i,j,k) ] = NRk[ DMAP(x) ].x * x[0] + NRk[ DMAP(x) ].y * x[1] + NRk[ DMAP(x) ].z;
               cube_wk[ POS3(i,j,k) ] = weightOweight * cube_wk[ POS3(i,j,k) ] / lambd / Rk[DMAP(x)];
               //F_k(p) W_k(p)
               cube[ POS3(i,j,k) ] = cube_w[ POS3(i,j,k) ] * cube[ POS3(i,j,k) ] + cube_wk[ POS3(i,j,k) ] * cube_k[ POS3(i,j,k) ];
               cube_w[ POS3(i,j,k) ] = min( cube_w[ POS3(i,j,k) ]+cube_wk[ POS3(i,j,k)], W_eta);
               cube[ POS3(i,j,k) ] /= cube_w[ POS3(i,j,k) ]+cube_wk[ POS3(i,j,k)];
            }
        */          
}


 
 
 
 /**
 * Host main routine
 */
 
int 
main(void)
{
   //---------------------------------------------------------------------------
   // Input
   //---------------------------------------------------------------------------
   //T_g,k
   float Rgk[9]={0.99957997,-0.003303149,-0.01823758,-0.0149635,1.0000104,0.010601612,0.018084152,-0.010273285,0.99978584};
   float tgk[3]={10.8916,11.8622,-9.30357};

   //--R and N_Rk
   float *Rk=(float *)malloc(map_size_x*map_size_y*sizeof(float));
   vertex *NRk=(vertex *)malloc(map_size_x*map_size_y*sizeof(vertex));
   
   
   
   //---------------------------------------------------------------------------
   // Allocation
   //---------------------------------------------------------------------------
   size_t size = size_x*size_y*size_z*sizeof(float);
   // Allocat *cube* vectors on host --may not malloc???
   //Fk
   float *cube=(float *)malloc(size);

   //Wk
   float *cube_w=(float *)malloc(size);

   //F_Rk
   float *cube_k=(float *)malloc(size);//???????????????????????????? No use

   //W_Rk
   float *cube_wk=(float *)malloc(size);

   // Verify that allocations succeeded
   if (cube == NULL || cube_w == NULL || cube_k == NULL || cube_wk == NULL)
   {
       fprintf(stderr, "Failed to allocate host vectors!\n");
       exit(EXIT_FAILURE);
   }


   //---------------------------------------------------------------------------
    // Allocate the device input vector cube   
   //---------------------------------------------------------------------------
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    float *d_cube = NULL;
    err = hipMalloc((void **)&d_cube, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector cube (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device input vector cube_w
    float *d_cube_w = NULL;
    err = hipMalloc((void **)&d_cube_w, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector cube_w (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device output vector cube_k
    float *d_cube_k = NULL;
    err = hipMalloc((void **)&d_cube_k, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector cube_k (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    // Allocate the device output vector cube_wk
    float *d_cube_wk = NULL;
    err = hipMalloc((void **)&d_cube_wk, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector cube_wk (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    // Allocate the device output vector Rgk
    float *d_Rgk = NULL;
    err = hipMalloc((void **)&d_Rgk, 9*sizeof(float));

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector Rgk (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    // Allocate the device output vector tgk
    float *d_tgk = NULL;
    err = hipMalloc((void **)&d_tgk, 3*sizeof(float));

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector tgk (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    // Allocate the device output vector Rk
    float *d_Rk = NULL;
    err = hipMalloc((void **)&d_Rk, map_size_x*map_size_y*sizeof(float));

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector Rk (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    // Allocate the device output vector NRk
    vertex *d_NRk = NULL;
    err = hipMalloc((void **)&d_NRk, map_size_x*map_size_y*sizeof(vertex));

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector Rk (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    //--------------------------------------------------------------------------
    // Copy the host input vectors in host memory to the device input vectors in
    // device memory
    //--------------------------------------------------------------------------    
    err = hipMemcpy(d_cube, cube, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector cube from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_cube_w, cube_w, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector cube_w from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    err = hipMemcpy(d_cube_k, cube_k, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector cube_k from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    err = hipMemcpy(d_cube_wk, cube_wk, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector cube_wk from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    err = hipMemcpy(d_Rgk, Rgk, 9*sizeof(float), hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector Rgk from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    

    err = hipMemcpy(d_tgk, tgk, 3*sizeof(float), hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector tgk from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    err = hipMemcpy(d_Rk, Rk, map_size_x*map_size_y*sizeof(float), hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector Rk from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    err = hipMemcpy(d_NRk, NRk, map_size_x*map_size_y*sizeof(vertex), hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector NRk from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    
    
    
    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =(size_x*size_y*size_z + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    TSDF<<<blocksPerGrid, threadsPerBlock>>>(d_cube, d_cube_w, d_cube_k, d_cube_wk, d_Rgk, d_tgk, d_Rk, d_NRk);
    err = hipGetLastError();
   // cube cube_w cube_k cube_wk Rgk tgk Rk NRk

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //--------------------------------------------------------------------------
    // Wait to be Kernel
    //--------------------------------------------------------------------------
   // The main loop ----------->kernel
   for (int i = 0 ; i<size_x ;i++)
      for (int j = 0 ; j<size_y ;j++)
         for (int k = 0 ; k<size_z ;k++)
         {
            //p---?? unit
            float p[3], pt[3];
            int x[2];
            p[0] = (i - center_x) * unit;
            p[1] = (j - center_y) *unit;
            p[2] = (k - center_z) *unit;
            //printf("p[2]=%f\n",p[2]);
            
            //T_gk^-1 * p
            pt[0]=p[0]-tgk[0];
            pt[1]=p[1]-tgk[1];
            pt[2]=p[2]-tgk[2];
            
            for ( int s=0 ; s<3 ; s++  )
               for ( int t=0 ; t<3 ; t++  )
               {
                  p[s] += Rgk[ POS(3,t,s) ] * pt[t];
               }
            
            //first part of eta
            float eta=sqrt(pt[0]*pt[0]+pt[1]*pt[1]+pt[2]*pt[2]);
            
            //pi( K * (T_gk^-1 * p) )
            //round to nearest pixel
            for ( int s=0 ; s<2 ; s++  )
              x[s]=int( p[s] + p[2] + 0.5f );
              
            if(-1 < x[0] < map_size_x && -1< x[1] < map_size_y)
            {
               //lambd=norm(K^-1*x)
               float lambd = 0;
               for ( int s=0 ; s<2 ; s++  )
                  lambd += x[s] * x[s];

               lambd=sqrt( lambd + 1.0f );
               

               //second part of eta
               eta=eta / lambd + Rk[DMAP(x)];
               
               //printf("cube[ POS3(i,j,k) ]=%f\n",cube_w[ POS3(i,j,k) ]+cube_wk[ POS3(i,j,k)]);
               //F_Rk(p)
               if (eta>=-mu)
                  cube_k[ POS3(i,j,k) ] = min(1.0f,eta/mu)*(-1);
               
               //W_Rk(p)
               cube_wk[ POS3(i,j,k) ] = NRk[ DMAP(x) ].x * x[0] + NRk[ DMAP(x) ].y * x[1] + NRk[ DMAP(x) ].z;
               cube_wk[ POS3(i,j,k) ] = weightOweight * cube_wk[ POS3(i,j,k) ] / lambd / Rk[DMAP(x)];
               //F_k(p) W_k(p)
               cube[ POS3(i,j,k) ] = cube_w[ POS3(i,j,k) ] * cube[ POS3(i,j,k) ] + cube_wk[ POS3(i,j,k) ] * cube_k[ POS3(i,j,k) ];
               cube_w[ POS3(i,j,k) ] = min( cube_w[ POS3(i,j,k) ]+cube_wk[ POS3(i,j,k)], W_eta);
               cube[ POS3(i,j,k) ] /= cube_w[ POS3(i,j,k) ]+cube_wk[ POS3(i,j,k)];
               
            }
         }









    

    //--------------------------------------------------------------------------
    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    //--------------------------------------------------------------------------
    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(cube, d_cube, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector cube from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    


    // Verify that the result vector is correct
    for (int i = 0; i < 10 ; ++i)
    {
      printf("cube[0]=%f\n",cube[0]);
    }
    

    // Free device global memory


    // Free host memory

    // Reset the device and exit
    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Done\n");
    return 0;
}


